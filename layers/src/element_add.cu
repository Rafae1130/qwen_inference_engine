#include<hip/hip_runtime.h>
#include<hip/hip_bf16.h>

__global__ void element_mul(__hip_bfloat16 *a, __hip_bfloat16 *b, __hip_bfloat16 *c, int size){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    float temp = 0;
    if(idx < size){
        temp =  __bfloat162float(a[idx]) * __bfloat162float(b[idx]);
        c[idx] = __float2bfloat16(temp);
    }
}