#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>

#include "layers_include.cuh"
#include "tensor_parser.hh"
#include <fstream>
#include <iostream>


#include <vector>
#include <algorithm>
#include <cstdio>
#include <hip/hip_bf16.h>


#include <iomanip>
#include <utils.hh>
#include <helpers.cuh>
#include "iengine.cuh"



#define PRINT_TIME

inline void startCudaTimer(hipEvent_t &start, hipEvent_t &stop) {
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
}

inline float stopCudaTimer(hipEvent_t &start, hipEvent_t &stop, const char* label, int layer) {
    float ms = 0.0f;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    if (label) {
        std::cout <<"Layer: "<< layer <<label << ": " << ms << " ms" << std::endl;
    } else {
        std::cout << "Elapsed time: " << ms << " ms" << std::endl;
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return ms;
}




int llm(batch_metadata *new_seq , std::unordered_map<std::string, std::vector<tensor>> tensors, std::ifstream &weights, page_table *kv_cache_seq1, int page_size){

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);


    ModelBuffers *buffer;
    buffer = new_seq->buffer;
    if (new_seq->state == prefill){


        for(size_t i = 0; i < buffer->number_of_layers; i++){

            std::cout << "loop iteration: " << i << std::endl;

            //normalization block
            load_weight(tensors["input_layernorm.weight"][i], weights, buffer->norm_weights_h, buffer->norm_weights_d, buffer->hidden_dim);
            launch_rms(buffer->embeddings_out, buffer->norm_weights_d, buffer->rms_out, buffer->hidden_dim, buffer->sequence_len);

            //projections - Q
            proj(tensors["self_attn.q_proj.weight"][i], weights,
                buffer->q_proj_weights_h, buffer->q_proj_weights_d, buffer->q_proj_size,
                buffer->rms_out, buffer->Q, buffer->sequence_len, 5120, 5120);    // Load q_proj weights (H->D) and compute Q = rms_out · W_q  [m=seqlen, n=hidden=5120, k=hidden=5120]
            //projections - K
            proj(tensors["self_attn.k_proj.weight"][i], weights,
                buffer->kv_proj_weights_h, buffer->kv_proj_weights_d, buffer->kv_proj_size,
                buffer->rms_out, buffer->K, buffer->sequence_len, 5120, 1024);    // Load k_proj weights (H->D) and compute K = rms_out · W_k  [m=seqlen, n=hidden=5120, k=kv=1024]
            //projections - V
            proj(tensors["self_attn.v_proj.weight"][i], weights,
                buffer->kv_proj_weights_h, buffer->kv_proj_weights_d, buffer->kv_proj_size,
                buffer->rms_out, buffer->V, buffer->sequence_len, 5120, 1024);    // Load v_proj weights (H->D) and compute V = rms_out · W_v  [m=seqlen, n=hidden=5120, k=kv=1024]

            // q_norm
            load_weight(tensors["self_attn.q_norm.weight"][i], weights, buffer->qk_norm_weights_h, buffer->qk_norm_weights_d, buffer->head_dim);
            launch_qknorm(buffer->Q, buffer->qk_norm_weights_d, buffer->head_dim, buffer->sequence_len, buffer->hidden_dim, buffer->num_of_qheads);

            // k_norm
            load_weight(tensors["self_attn.k_norm.weight"][i], weights, buffer->qk_norm_weights_h, buffer->qk_norm_weights_d, buffer->head_dim);
            launch_qknorm(buffer->K, buffer->qk_norm_weights_d, buffer->head_dim, buffer->sequence_len, buffer->hidden_dim_kv, buffer->num_of_kvheads);

            // RoPE
            launch_rope(buffer->cos_values_d, buffer->sin_values_d, buffer->Q, buffer->sequence_len, buffer->head_dim, buffer->hidden_dim, buffer->num_of_qheads); // Apply RoPE to Q for all query heads
            launch_rope(buffer->cos_values_d, buffer->sin_values_d, buffer->K, buffer->sequence_len, buffer->head_dim, buffer->hidden_dim_kv, buffer->num_of_kvheads); // Apply RoPE to K for all KV heads


        
            
            // #ifdef PRINT_TIME
            // float ms = stopCudaTimer(start, stop, "Time in kv_cache copy", i);
            // #endif

            const int tokens_per_page = page_size;                          // <-- tokens per page
            const size_t vec_elems     = (size_t)buffer->hidden_dim_kv;
            const size_t pitch_bytes   = vec_elems * sizeof(__hip_bfloat16);
            const size_t width_bytes   = pitch_bytes;                 // copy a full token vector per row

            int sequence_len = buffer->sequence_len;

            #ifdef PRINT_TIME
            startCudaTimer(start, stop);
            #endif

            if (sequence_len <= tokens_per_page) {
                // Fast path: fits in the first (flat) cache or first page
                // If you still want to keep the old flat cache for small seqs:
                // dpitch/spitch are both the per-token byte span
                const size_t kv_dim      = buffer->hidden_dim_kv;
                const size_t dpitch = (size_t)buffer->number_of_layers * kv_dim * sizeof(__hip_bfloat16);;
                const size_t spitch = kv_dim * sizeof(__hip_bfloat16);;
                const int height    = sequence_len;                   // rows = tokens

                // Copy K
                hipMemcpy2D(
                    /* dst    */ &buffer->k_cache[i * buffer->hidden_dim_kv], // if you still maintain a flat cache
                    /* dpitch */ dpitch,
                    /* src    */ buffer->K,
                    /* spitch */ spitch,
                    /* width  */ width_bytes,
                    /* height */ height,
                    /* kind   */ hipMemcpyDeviceToDevice
                );

                // Copy V
                hipMemcpy2D(
                    &buffer->v_cache[i * buffer->hidden_dim_kv],
                    dpitch,
                    buffer->V,
                    spitch,
                    width_bytes,
                    height,
                    hipMemcpyDeviceToDevice
                );
            } else {
                // Paged path
                // How many pages do we need?
                const int pages_required = (sequence_len + tokens_per_page - 1) / tokens_per_page;

                page_table* temp = kv_cache_seq1; // head page
                if (!temp) { /* handle null / allocate first page */ }

                int copied_tokens = 0;
                for (int k = 0; k < pages_required; k++) {
                    if (!temp) {
                        // You ran out of pre-allocated pages — allocate or error out
                        // allocate_next_page(&temp, tokens_per_page * vec_elems);
                        // or: throw / return
                        break;
                    }

                    // How many tokens to write into this page
                    int remaining      = sequence_len - copied_tokens;
                    int this_page_rows = (remaining < tokens_per_page) ? remaining : tokens_per_page;

                    const size_t kv_dim      = buffer->hidden_dim_kv;                // 1024
                    const size_t src_pitch   = kv_dim * sizeof(__hip_bfloat16);       // row = kv vector
                    const size_t dst_pitch   = (size_t)buffer->number_of_layers * kv_dim * sizeof(__hip_bfloat16); // jump over all layers per token
                    const size_t width_bytes = src_pitch;

                  
                    // K
                    hipMemcpy2D(
                        /* dst    */ temp->k_page_ptr + (size_t)i * kv_dim,  // start at this layer inside the [layer] slab
                        /* dpitch */ dst_pitch,                               // next token jumps over all layers*kv
                        /* src    */ buffer->K + (size_t)copied_tokens * kv_dim,   // source token t base
                        /* spitch */ src_pitch,                               // next token in src is just +kv
                        /* width  */ width_bytes,                             // copy one kv vector
                        /* height */ this_page_rows,                          // number of tokens in this page
                        /* kind   */ hipMemcpyDeviceToDevice
                    );

                    // V
                    hipMemcpy2D(
                        temp->v_page_ptr + (size_t)i * kv_dim,
                        dst_pitch,
                        buffer->V + (size_t)copied_tokens * kv_dim,
                        src_pitch,
                        width_bytes,
                        this_page_rows,
                        hipMemcpyDeviceToDevice
                    );

                    // Bookkeeping
                    temp->page_allocated = this_page_rows;  // optional
                    copied_tokens += this_page_rows;
                    temp = temp->ptr_to_next_page;
                }
            }

            #ifdef PRINT_TIME
            stopCudaTimer(start, stop, "KV paged copy", i);
            #endif


            //self attention
            #ifdef PRINT_TIME
            startCudaTimer(start, stop);
            #endif
            launch_attn(buffer->Q, buffer->k_cache, buffer->v_cache, buffer->atten_out, buffer->sequence_len, buffer->sequence_len, buffer->head_dim, buffer->hidden_dim, buffer->hidden_dim_kv, /*causal=*/1, 0, i, kv_cache_seq1, page_size); // self-attention (prefill uses full causal window)
            #ifdef PRINT_TIME
            stopCudaTimer(start, stop, "Time in attention", i);
            #endif
            
            
            
            
            
            //outut projection self attention
            #ifdef PRINT_TIME
            startCudaTimer(start, stop);
            #endif
            proj(tensors["self_attn.o_proj.weight"][i], weights, buffer->o_proj_weights_h, buffer->o_proj_weights_d, buffer->o_proj_size, buffer->atten_out, buffer->out_proj, buffer->sequence_len, /*n=*/5120, /*k=*/5120); // output projection (O proj)
            #ifdef PRINT_TIME

            stopCudaTimer(start, stop, "Time in o_projction", i);
            #endif

            //residual add
            #ifdef PRINT_TIME
            startCudaTimer(start, stop);
            #endif
            launch_resadd(buffer->embeddings_out, buffer->out_proj, buffer->sequence_len * buffer->hidden_dim); // residual add: x += out_proj
            #ifdef PRINT_TIME

            stopCudaTimer(start, stop, "Time in residual add", i);
            #endif

            //normalization
            #ifdef PRINT_TIME
            startCudaTimer(start, stop);
            #endif
            load_weight(tensors["post_attention_layernorm.weight"][i], weights, buffer->norm_weights_h, buffer->norm_weights_d, buffer->hidden_dim); // load post-attn RMSNorm weights to device
            launch_rms(buffer->embeddings_out, buffer->norm_weights_d, buffer->rms_out, buffer->hidden_dim, buffer->sequence_len); // post-attention RMSNorm
            
            #ifdef PRINT_TIME

            stopCudaTimer(start, stop, "Time in rms", i);
            #endif

            //MLP
            #ifdef PRINT_TIME
            startCudaTimer(start, stop);
            #endif
            proj(tensors["mlp.up_proj.weight"][i],   weights, buffer->mlp_up_proj_weights_h, buffer->mlp_up_proj_weights_d, buffer->mlp_up_proj_size, buffer->rms_out,      buffer->MLP_UP,        buffer->sequence_len, /*n=*/5120,           /*k=*/buffer->up_dim); // MLP up-proj
            #ifdef PRINT_TIME

            stopCudaTimer(start, stop, "Time in MLP up proj", i);
            #endif


            #ifdef PRINT_TIME
            startCudaTimer(start, stop);
            #endif
            proj(tensors["mlp.gate_proj.weight"][i], weights, buffer->mlp_up_proj_weights_h, buffer->mlp_up_proj_weights_d, buffer->mlp_up_proj_size, buffer->rms_out,      buffer->MLP_GATE,      buffer->sequence_len, /*n=*/5120,           /*k=*/buffer->up_dim); // MLP gate-proj
            #ifdef PRINT_TIME

            stopCudaTimer(start, stop, "Time in MLP  gate proj", i);
            #endif

            #ifdef PRINT_TIME
            startCudaTimer(start, stop);
            #endif
            launch_act(buffer->MLP_GATE, buffer->sequence_len * buffer->up_dim); // activation (e.g., SiLU) on gate
            launch_elem(buffer->MLP_UP, buffer->MLP_GATE, buffer->MLP_GATE_OUT, buffer->sequence_len * buffer->up_dim); // elementwise multiply: up * act(gate)
            #ifdef PRINT_TIME

            stopCudaTimer(start, stop, "Time in activation and multiply", i);
            #endif

            #ifdef PRINT_TIME
            startCudaTimer(start, stop);
            #endif
            proj(tensors["mlp.down_proj.weight"][i], weights, buffer->mlp_up_proj_weights_h, buffer->mlp_up_proj_weights_d, buffer->mlp_up_proj_size, buffer->MLP_GATE_OUT, buffer->MLP_DOWN,      buffer->sequence_len, /*n=*/buffer->up_dim,  /*k=*/buffer->hidden_dim); // MLP down-proj
            #ifdef PRINT_TIME

            stopCudaTimer(start, stop, "Time in down projection", i);
            #endif


            #ifdef PRINT_TIME
            startCudaTimer(start, stop);
            #endif
            launch_resadd(buffer->embeddings_out, buffer->MLP_DOWN, buffer->sequence_len * buffer->hidden_dim); // residual add: x += mlp_down
            #ifdef PRINT_TIME

            stopCudaTimer(start, stop, "Time in final residual add", i);
            #endif
        }


        // final RMSNorm before logits
        load_weight(tensors["norm.weight"][0], weights, buffer->norm_weights_h, buffer->norm_weights_d, buffer->hidden_dim);  // load γ
        launch_rms (buffer->embeddings_out, buffer->norm_weights_d, buffer->rms_out, buffer->hidden_dim, buffer->sequence_len); // y = rmsnorm(x)

        // load logits (W_vocab)
        load_weight(tensors["logits"][0], weights, buffer->logits_weights_h, buffer->logits_weights_d, buffer->logtis_shape);          

        // pick last token activation
        copy_last_vocab_vec(buffer->rms_out, buffer->last_x, buffer->hidden_dim, buffer->sequence_len);                                            

        // logits = last_x · W_vocab
        launch_matmul(buffer->last_x, buffer->logits_weights_d, buffer->prefill_output_d, /*m=*/1, /*n=*/buffer->hidden_dim, /*k=*/buffer->vocab_size); 


        //output sampling
        int step = 0;                                                                                                      // decode step counter
        int out_logit = sample_topk_bf16(buffer->prefill_output_d, buffer->vocab_size, /*temperature=*/1.0f, /*topk=*/50,
                                        /*seed=*/1234ULL, /*step=*/step);                                                 // sample next token id
        // std::cout << "out_logit: " << out_logit << std::endl;  


        return out_logit;
    }


    else if (new_seq->state == decode){
        int out_logit = new_seq->generated_token;
        // int seq = buffer->sequence_len;
        int *d_token_ids_decode;
        hipMalloc((void **)&d_token_ids_decode, sizeof(int));
        
        //decode stage
        while(out_logit != 151645){
            int step = new_seq->step;
            int threads = 1;
            int blocks = 1;
            hipMemcpy(d_token_ids_decode, &out_logit, sizeof(int), hipMemcpyHostToDevice);
            hipError_t e = hipGetLastError(); if (e != hipSuccess) { fprintf(stderr, "hipMemcpy before embedding decode: %s\n", hipGetErrorString(e)); return 0; } hipDeviceSynchronize();

            size_t sequence_len_q = 1;
            buffer->sequence_len = buffer->sequence_len + 1;

            embedding_matrix_func<<<blocks,threads>>>(buffer->embeddings_out, buffer->embeddings_d, d_token_ids_decode, buffer->hidden_dim, sequence_len_q);
            e = hipGetLastError(); if (e != hipSuccess) { fprintf(stderr, "Kernel embedding decode: %s\n", hipGetErrorString(e)); return 0; } hipDeviceSynchronize();


            for(size_t  i =0; i < buffer->number_of_layers; i++){
                // std::cout << "loop iteration: " << i << std::endl;
                // rms norm (input LN for decode token)
                load_weight(tensors["input_layernorm.weight"][i], weights, buffer->norm_weights_h, buffer->norm_weights_d, buffer->hidden_dim);
                launch_rms(buffer->embeddings_out, buffer->norm_weights_d, buffer->rms_out, buffer->hidden_dim, sequence_len_q);
                e = hipGetLastError(); if (e != hipSuccess) { fprintf(stderr, "Kernel rmsNorm decode: %s\n", hipGetErrorString(e)); return 0; } hipDeviceSynchronize();

                // Q projection
                proj(tensors["self_attn.q_proj.weight"][i], weights, buffer->q_proj_weights_h, buffer->q_proj_weights_d, buffer->q_proj_size,
                    /*x=*/buffer->rms_out, /*y=*/buffer->Q, /*m=*/sequence_len_q, /*n=*/5120, /*k=*/5120);
                e = hipGetLastError(); if (e != hipSuccess) { fprintf(stderr, "Kernel matmul: %s\n", hipGetErrorString(e)); return 0; } hipDeviceSynchronize();

                // K projection
                proj(tensors["self_attn.k_proj.weight"][i], weights, buffer->kv_proj_weights_h, buffer->kv_proj_weights_d, buffer->kv_proj_size,
                    /*x=*/buffer->rms_out, /*y=*/buffer->K, /*m=*/sequence_len_q, /*n=*/5120, /*k=*/1024);

                // V projection
                proj(tensors["self_attn.v_proj.weight"][i], weights, buffer->kv_proj_weights_h, buffer->kv_proj_weights_d, buffer->kv_proj_size,
                    /*x=*/buffer->rms_out, /*y=*/buffer->V, /*m=*/sequence_len_q, /*n=*/5120, /*k=*/1024);

                // q_norm
                load_weight(tensors["self_attn.q_norm.weight"][i], weights, buffer->qk_norm_weights_h, buffer->qk_norm_weights_d, buffer->head_dim);
                launch_qknorm(buffer->Q, buffer->qk_norm_weights_d, buffer->head_dim, sequence_len_q, buffer->hidden_dim, buffer->num_of_qheads);

                // k_norm
                load_weight(tensors["self_attn.k_norm.weight"][i], weights, buffer->qk_norm_weights_h, buffer->qk_norm_weights_d, buffer->head_dim);
                launch_qknorm(buffer->K, buffer->qk_norm_weights_d, buffer->head_dim, sequence_len_q, buffer->hidden_dim_kv, buffer->num_of_kvheads);

                // RoPE (single position = buffer->sequence_len - 1)
                launch_rope_single(buffer->cos_values_d, buffer->sin_values_d, buffer->Q, buffer->sequence_len - 1, buffer->head_dim, buffer->hidden_dim,    buffer->num_of_qheads);
                e = hipGetLastError(); if (e != hipSuccess) { fprintf(stderr, "Kernel rope: %s\n", hipGetErrorString(e)); return 0; } hipDeviceSynchronize();
                launch_rope_single(buffer->cos_values_d, buffer->sin_values_d, buffer->K, buffer->sequence_len - 1, buffer->head_dim, buffer->hidden_dim_kv, buffer->num_of_kvheads);
                e = hipGetLastError(); if (e != hipSuccess) { fprintf(stderr, "Kernel rope: %s\n", hipGetErrorString(e)); return 0; } hipDeviceSynchronize();
            


                // //kv cache
                // size_t pos = buffer->sequence_len - 1;                  
                // size_t layer_off = size_t(i) + buffer->context_size*buffer->hidden_dim_kv;
                // size_t dst_off   = layer_off + size_t(pos)*buffer->hidden_dim_kv;

                // hipMemcpy(&buffer->k_cache[dst_off], buffer->K, buffer->hidden_dim_kv*sizeof(__hip_bfloat16), hipMemcpyDeviceToDevice);
                // hipMemcpy(&buffer->v_cache[dst_off], buffer->V, buffer->hidden_dim_kv*sizeof(__hip_bfloat16), hipMemcpyDeviceToDevice);

                #ifdef PRINT_TIME
                startCudaTimer(start, stop);
                #endif

                size_t pos = buffer->sequence_len - 1;


                int page_idx = pos / page_size;
                int offset_in_page = pos % page_size;

                // walk to the correct page
                page_table* page = kv_cache_seq1;
                for (int p = 0; p < page_idx && page; ++p)
                    page = page->ptr_to_next_page;
                if (!page) {
                    fprintf(stderr, "Error: Page %d not allocated (pos=%zu, page_size=%d)\n", page_idx, pos, page_size);
                    page_table* new_page = kv_cache_seq1;
                    for (int p = 0; p < page_idx -1; ++p)
                        new_page = new_page->ptr_to_next_page;
                    new_page->ptr_to_next_page = create_page_list(1);
                    page = new_page->ptr_to_next_page;
                    int elements_per_page =  page_size * buffer->number_of_layers * (size_t)buffer->hidden_dim_kv;;
                    allocate_page_buffers(page, elements_per_page);
                    if(page){
                        std::cout << "new page allocated" << std::endl;
                    }

                }



                // Safety check
                if (!page) {
                    fprintf(stderr, "Critical error: Failed to allocate/access page\n");
                    return 0;
                }

                // compute offset inside that page
                size_t layer_off = (size_t)offset_in_page * buffer->number_of_layers * buffer->hidden_dim_kv  // jump to token
                                + (size_t)i * buffer->hidden_dim_kv;   

                std::cout <<"before cache copy in decode" << std::endl;
                // write into the correct page’s memory
                hipMemcpy(&page->k_page_ptr[layer_off], buffer->K,
                        buffer->hidden_dim_kv * sizeof(__hip_bfloat16),
                        hipMemcpyDeviceToDevice);

                hipMemcpy(&page->v_page_ptr[layer_off], buffer->V,
                        buffer->hidden_dim_kv * sizeof(__hip_bfloat16),
                        hipMemcpyDeviceToDevice);

                #ifdef PRINT_TIME
                float ms = stopCudaTimer(start, stop, "Layer copy (paged)", i);
                #endif

                hipDeviceSynchronize();
                    
                #ifdef PRINT_TIME
                    startCudaTimer(start, stop);
                #endif
                // self‑attention 
                int q_abs = buffer->sequence_len - 1;
                // launch_attn(buffer->Q, &buffer->k_cache[i*(buffer->context_size*buffer->hidden_dim_kv)], &buffer->v_cache[i*(buffer->context_size*buffer->hidden_dim_kv)], buffer->atten_out, /*mq=*/sequence_len_q, /*mkv=*/buffer->sequence_len, /*head_dim=*/buffer->head_dim, /*hidden=*/buffer->hidden_dim, /*hidden_kv=*/buffer->hidden_dim_kv, /*causal=*/0, q_abs, i); // SA(Q, Kcache, Vcache) → atten_out
                launch_attn(buffer->Q, buffer->k_cache, buffer->v_cache, buffer->atten_out, /*mq=*/sequence_len_q, /*mkv=*/buffer->sequence_len, /*head_dim=*/buffer->head_dim, /*hidden=*/buffer->hidden_dim, /*hidden_kv=*/buffer->hidden_dim_kv, /*causal=*/0, q_abs, i, kv_cache_seq1, page_size); // SA(Q, Kcache, Vcache) → atten_out

                #ifdef PRINT_TIME
                    hipDeviceSynchronize();
                    ms = stopCudaTimer(start, stop, "Time in attention", i);
                #endif

                // output projection 
                proj(tensors["self_attn.o_proj.weight"][i], weights, buffer->o_proj_weights_h, buffer->o_proj_weights_d, buffer->o_proj_size, /*x=*/buffer->atten_out, /*y=*/buffer->out_proj, /*m=*/sequence_len_q, /*n=*/5120, /*k=*/5120); // O-proj

                // residual add
                launch_resadd(buffer->embeddings_out, buffer->out_proj, /*n=*/sequence_len_q * buffer->hidden_dim); // add SA output

                // post-attention RMSNorm
                load_weight(tensors["post_attention_layernorm.weight"][i], weights, buffer->norm_weights_h, buffer->norm_weights_d, buffer->hidden_dim); // load LN2 weights
                launch_rms(buffer->embeddings_out, buffer->norm_weights_d, buffer->rms_out, buffer->hidden_dim, sequence_len_q); // LN2(x) → rms_out

                // MLP up-proj
                proj(tensors["mlp.up_proj.weight"][i],   weights, buffer->mlp_up_proj_weights_h, buffer->mlp_up_proj_weights_d, buffer->mlp_up_proj_size, /*x=*/buffer->rms_out, /*y=*/buffer->MLP_UP,   /*m=*/sequence_len_q, /*n=*/5120, /*k=*/buffer->up_dim); // up-proj

                // MLP gate-proj
                proj(tensors["mlp.gate_proj.weight"][i], weights, buffer->mlp_up_proj_weights_h, buffer->mlp_up_proj_weights_d, buffer->mlp_up_proj_size, /*x=*/buffer->rms_out, /*y=*/buffer->MLP_GATE, /*m=*/sequence_len_q, /*n=*/5120, /*k=*/buffer->up_dim); // gate-proj

                // activation on gate
                launch_act(buffer->MLP_GATE, /*n=*/sequence_len_q * buffer->up_dim); // act(G) in-place

                // gated product
                launch_elem(buffer->MLP_UP, buffer->MLP_GATE, buffer->MLP_GATE_OUT, /*n=*/sequence_len_q * buffer->up_dim); // U * act(G)

                // MLP down-proj
                proj(tensors["mlp.down_proj.weight"][i], weights, buffer->mlp_up_proj_weights_h, buffer->mlp_up_proj_weights_d, buffer->mlp_up_proj_size, /*x=*/buffer->MLP_GATE_OUT, /*y=*/buffer->MLP_DOWN, /*m=*/sequence_len_q, /*n=*/buffer->up_dim, /*k=*/buffer->hidden_dim); // down-proj

                // residual add
                launch_resadd(buffer->embeddings_out, buffer->MLP_DOWN, /*n=*/sequence_len_q * buffer->hidden_dim); // add MLP output

            }

            // final token RMSNorm
            load_weight(tensors["norm.weight"][0], weights, buffer->norm_weights_h, buffer->norm_weights_d, buffer->hidden_dim);                   // γ
            launch_rms(buffer->embeddings_out, buffer->norm_weights_d, buffer->rms_out, buffer->hidden_dim, /*seqlen=*/sequence_len_q);            // y = rmsnorm(x)

            // load logits matrix and copy first token state
            load_weight(tensors["logits"][0],     weights, buffer->logits_weights_h, buffer->logits_weights_d, buffer->logtis_shape);             // W_vocab
            copy_first_token(buffer->rms_out, buffer->last_x, buffer->hidden_dim);                                                                 // last_x ← rms_out[0]

            // logits 
            launch_matmul(buffer->last_x, buffer->logits_weights_d, buffer->prefill_output_d, /*m=*/1, /*n=*/buffer->hidden_dim, /*k=*/buffer->vocab_size);

            e = hipGetLastError();
            if (e != hipSuccess) { fprintf(stderr, "Kernel matrix_mul: decode %s\n", hipGetErrorString(e)); return 0; }
            hipDeviceSynchronize();


            //output sampling
            int* d_output_token;
            float temperature = 0.7;
            int topk = 50;
            hipMalloc(&d_output_token, sizeof(int));
            hipDeviceSynchronize();

            // std::cout<<"going in logits_decode kernel"<< std::endl;
            
            topk_temperature_softmax_sampling_kernel_bf16<<<1, 256>>>(buffer->prefill_output_d, d_output_token,temperature,topk,buffer->vocab_size,1234 + step, 0 );

            e = hipGetLastError();
            if (e != hipSuccess) { fprintf(stderr, "Kernel topk_temperature_softmax_sampling_kernel_bf16: decode %s\n", hipGetErrorString(e)); return 0; }
            hipDeviceSynchronize();
            
            int out_logit = 0;
            
            hipMemcpy(&out_logit, d_output_token, sizeof(int), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();

            // std::cout << "out_logit: "<< out_logit<< std::endl;
            hipFree(d_output_token);
            
            hipFree(d_token_ids_decode);
            return out_logit;   
        }   


    }
    hipDeviceSynchronize();
 
    // if (weights.is_open()) weights.close();
    // destroy_model_buffers(buffer);
    std::cout <<"Reached end"<<std::endl;



    return 0;
}







